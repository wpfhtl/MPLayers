#include "cudaUtils.hpp"

void __CUDAErrorCheck(const char *file,
                      const int line) {
    hipError_t status = hipGetLastError();
    if (hipSuccess != status) {
        printf("!!!Error, CUDAErrorCheck failed, %s:%i : %s\n",
               file, line, hipGetErrorString(status));
        exit(-1);
    }

    status = hipDeviceSynchronize();
    if(hipSuccess != status) {
        printf("!!!Error, CUDAErrorCheck sync failed, %s:%i : %s\n",
               file, line, hipGetErrorString(status));
        exit(-1);
    }
}
